#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>

// Error checking macro
#define CHECK_CUDA(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        const char* errStr; \
        hipDrvGetErrorString(err, &errStr); \
        printf("CUDA Error: %s at %s:%d\n", errStr, __FILE__, __LINE__); \
        return -1; \
    } \
} while(0)

// Declaration of the PTX kernel
extern "C" __global__ void wmma_kernel(
    half* a,    // Input matrix A (fp16)
    half* b,    // Input matrix B (fp16)
    float* c,   // Input accumulator matrix C (fp32)
    float* d    // Output matrix D (fp32)
);

// Helper function to initialize matrices
void init_matrices(half* a, half* b, float* c, int m, int n, int k) {
    // Initialize A (m x k)
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            a[i * k + j] = __float2half(1.0f);  // All ones for simplicity
        }
    }
    
    // Initialize B (k x n)
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < n; j++) {
            b[i * n + j] = __float2half(1.0f);  // All ones for simplicity
        }
    }
    
    // Initialize C (m x n)
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            c[i * n + j] = 0.0f;  // Zero initialization
        }
    }
}

// Helper function to verify results
void verify_results(float* d, int m, int n, int k) {
    bool passed = true;
    // For A and B filled with 1.0, each element in D should be k (dot product of k ones)
    float expected = (float)k;  // k ones multiplied together
    
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (fabs(d[i * n + j] - expected) > 1e-3) {
                printf("Mismatch at [%d,%d]: expected %.2f, got %.2f\n", 
                       i, j, expected, d[i * n + j]);
                passed = false;
            }
        }
    }
    printf("Test %s\n", passed ? "PASSED" : "FAILED");
    if (passed) {
        printf("All elements correctly computed as %.2f (dot product of %d ones)\n", 
               expected, k);
    }
}

int main() {
    const int M = 16;
    const int N = 16;
    const int K = 16;
    const int size_a = M * K;
    const int size_b = K * N;
    const int size_c = M * N;
    const int size_d = M * N;

    // Initialize CUDA driver API
    CHECK_CUDA(hipInit(0));

    // Get a CUDA device
    hipDevice_t device;
    CHECK_CUDA(hipDeviceGet(&device, 0));

    // Create context
    hipCtx_t context;
    CHECK_CUDA(hipCtxCreate(&context, 0, device));

    // Load PTX module from file
    hipModule_t module;
    CHECK_CUDA(hipModuleLoad(&module, "wmma_kernel_demo.ptx"));

    // Get kernel function
    hipFunction_t kernel;
    CHECK_CUDA(hipModuleGetFunction(&kernel, module, "wmma_kernel"));

    // Host memory
    half *h_a = new half[size_a];
    half *h_b = new half[size_b];
    float *h_c = new float[size_c];
    float *h_d = new float[size_d];

    // Initialize matrices with simpler values
    init_matrices(h_a, h_b, h_c, M, N, K);

    // Device memory
    hipDeviceptr_t d_a, d_b, d_c, d_d;
    CHECK_CUDA(hipMalloc(&d_a, size_a * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_b, size_b * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_c, size_c * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_d, size_d * sizeof(float)));

    // Copy data to device
    CHECK_CUDA(hipMemcpyHtoD(d_a, h_a, size_a * sizeof(half)));
    CHECK_CUDA(hipMemcpyHtoD(d_b, h_b, size_b * sizeof(half)));
    CHECK_CUDA(hipMemcpyHtoD(d_c, h_c, size_c * sizeof(float)));

    // Set up kernel parameters
    void* params[] = {
        &d_a,
        &d_b,
        &d_c,
        &d_d
    };

    // Launch kernel
    CHECK_CUDA(hipModuleLaunchKernel(kernel,
        1, 1, 1,    // Grid dimensions
        32, 1, 1,   // Block dimensions (one warp)
        0, NULL,    // Shared memory and stream
        params,     // Parameters
        NULL        // Extra (unused)
    ));

    // Copy result back to host
    CHECK_CUDA(hipMemcpyDtoH(h_d, d_d, size_d * sizeof(float)));

    // Verify results with corrected expectations
    verify_results(h_d, M, N, K);

    // Cleanup
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    delete[] h_d;
    CHECK_CUDA(hipFree(d_a));
    CHECK_CUDA(hipFree(d_b));
    CHECK_CUDA(hipFree(d_c));
    CHECK_CUDA(hipFree(d_d));
    CHECK_CUDA(hipModuleUnload(module));
    CHECK_CUDA(hipCtxDestroy(context));

    return 0;
} 